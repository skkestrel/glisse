#include <iomanip>
#include <fstream>
#include <unordered_map>
#include <algorithm>

#include "types.h"
#include "executor.cuh"
#include "wh.cuh"
#include "convert.h"
#include "wh.h"

ExecutorData::ExecutorData() { }
ExecutorData::ExecutorData(size_t n)
{
	r = v = std::vector<f64_3>(n);
	id = deathtime_index = std::vector<uint32_t>(n);
	deathflags = std::vector<uint16_t>(n);
}

template<typename T>
hipError_t memcpy_dth(std::vector<T>& dest, const thrust::device_vector<T>& src, hipStream_t& stream, size_t destbegin = 0, size_t srcbegin = 0, size_t len = static_cast<uint32_t>(-1))
{
	if (len == static_cast<uint32_t>(-1))
	{
		len = src.size();
	}
	if (dest.size() < destbegin + len)
	{
		throw std::exception();
	}

	return hipMemcpyAsync(dest.data() + destbegin, src.data().get() + srcbegin, len * sizeof(T), hipMemcpyDeviceToHost, stream);
}

template<typename T>
hipError_t memcpy_htd(thrust::device_vector<T>& dest, const std::vector<T>& src, hipStream_t& stream, size_t destbegin = 0, size_t srcbegin = 0, size_t len = static_cast<uint32_t>(-1))
{
	if (len == static_cast<uint32_t>(-1))
	{
		len = src.size();
	}
	if (dest.size() < destbegin + len)
	{
		throw std::exception();
	}

	return hipMemcpyAsync(dest.data().get() + destbegin, src.data() + srcbegin, len * sizeof(T), hipMemcpyHostToDevice, stream);
}

struct DeviceParticleUnflaggedPredicate
{
	template<typename Tuple>
	__host__ __device__
	bool operator()(const Tuple& args)
	{
		uint8_t flag = thrust::get<3>(args);
		return flag == 0;
	}
};

Executor::Executor(HostData& hd, DeviceData& dd, std::ostream& out) : hd(hd), dd(dd), print_every(10), print_counter(0), tbsize(128), ce_factor(1),
	output(out), timing_output(nullptr), resolve_encounters(false) { }

void Executor::init()
{
	to_helio(hd);

	calculate_planet_metrics(hd.planets, &e_0, nullptr);

	output << std::setprecision(7);
	output << "e_0 (planets) = " << e_0 << std::endl;
	output << "n_particle = " << hd.particles.n << std::endl;
	output << "n_particle_alive = " << hd.particles.n_alive << std::endl;
	output << "==================================" << std::endl;
	output << "Running for half a time step.     " << std::endl;

	initialize(hd.planets, hd.particles);

	output << "==================================" << std::endl;
	output << "Sending initial conditions to GPU." << std::endl;

	hipStreamCreate(&main_stream);
	hipStreamCreate(&htd_stream);
	hipStreamCreate(&par_stream);
	hipStreamCreate(&dth_stream);

	upload_data();
	output << "n_particle_alive = " << dd.particle_phase_space().n_alive << std::endl;

	resync();
	download_data();

	starttime = std::chrono::high_resolution_clock::now();

	output << "       Starting simulation.       " << std::endl << std::endl;

	if (timing_output)
	{
		*timing_output << std::setprecision(17);
	}
	if (discard_output)
	{
		*discard_output << std::setprecision(17);
	}

	step_and_upload_planets();
	if (!resolve_encounters)
	{
		ce_factor = 1;
	}
}

void Executor::step_and_upload_planets()
{
	if (resolve_encounters)
	{

		for (size_t i = 0; i < tbsize * ce_factor; i++)
		{
			step_planets(hd.planets, t, i, dt / ce_factor);
			// take the planet positions at the end of every timestep

			if (i % ce_factor == ce_factor - 1)
			{
				size_t slow_index = i / ce_factor;
				auto fast_begin = hd.planets.r_log.begin() + i * (hd.planets.n - 1);
				std::copy(fast_begin, fast_begin + (hd.planets.n - 1), hd.planets.r_log_slow.begin() + slow_index * (hd.planets.n - 1));

				hd.planets.h0_log_slow[i / ce_factor] = hd.planets.h0_log[i];
			}
		}
	}
	else
	{
		for (size_t i = 0; i < tbsize; i++)
		{
			step_planets(hd.planets, t, i, dt);
		}

		std::copy(hd.planets.h0_log.begin(), hd.planets.h0_log.end(), hd.planets.h0_log_slow.begin());
		std::copy(hd.planets.r_log.begin(), hd.planets.r_log.end(), hd.planets.r_log_slow.begin());
	}

	upload_planet_log();
}

void Executor::upload_data()
{
	dd.particles0 = DeviceParticlePhaseSpace(hd.particles.n);
	dd.particles1 = DeviceParticlePhaseSpace(hd.particles.n);

	dd.planets0 = DevicePlanetPhaseSpace(hd.planets.n, tbsize);
	dd.planets1 = DevicePlanetPhaseSpace(hd.planets.n, tbsize);

	dd.planet_data_id = 0;
	dd.particle_data_id = 0;

	auto& particles = dd.particle_phase_space();

	particles.n_alive = hd.particles.n_alive;

	memcpy_htd(particles.r, hd.particles.r, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.v, hd.particles.v, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.a, hd.particles.a, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.deathflags, hd.particles.deathflags, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.id, hd.particles.id, htd_stream);
	hipStreamSynchronize(htd_stream);

	memcpy_htd(dd.planet_phase_space().m, hd.planets.m, htd_stream);
	hipStreamSynchronize(htd_stream);

	dd.planet_data_id++;
	memcpy_htd(dd.planet_phase_space().m, hd.planets.m, htd_stream);
	hipStreamSynchronize(htd_stream);
}

void Executor::add_job(const std::function<void()>& job)
{
	work.push_back(std::move(job));
}

void Executor::download_data()
{
	auto& particles = dd.particle_phase_space();

	Vu32 prev_ids(hd.particles.id.begin(), hd.particles.id.end());

	memcpy_dth(hd.particles.r, particles.r, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.v, particles.v, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.a, particles.a, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.id, particles.id, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.deathflags, particles.deathflags, dth_stream);
	hipStreamSynchronize(dth_stream);

	if (prev_ids != hd.particles.id)
	{
		output << "WARNING! ID MISMATCH! WARNING!" << std::endl;
		throw std::exception();
	}

	hd.particles.n_alive = dd.particle_phase_space().n_alive;

	/*
	// zip will crash the program

	auto iterator = thrust::make_zip_iterator(thrust::make_tuple(
				ps.r.begin(),
				ps.v.begin(),
				ps.deathflags.begin(), ps.deathtime.begin(), ps.id.begin()));
	thrust::copy(thrust::cuda::par.on(stream),
			iterator,
			iterator + n,
			thrust::make_zip_iterator(thrust::make_tuple(
					hd.particles.r.begin(), hd.particles.v.begin(),
					hd.particles.deathflags.begin(), hd.deathtime.begin(), hd.id.begin())));
	 */
}

void Executor::upload_planet_log()
{
	dd.planet_data_id++;
	auto& planets = dd.planet_phase_space();

	memcpy_htd(planets.h0_log, hd.planets.h0_log_slow, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(planets.r_log, hd.planets.r_log_slow, htd_stream);
	hipStreamSynchronize(htd_stream);
}


double Executor::time() const
{
	auto now = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> millis = now - starttime;
	return millis.count() / 60000;
}

void Executor::loop()
{
	step_particles_cuda(main_stream, dd.planet_phase_space(), dd.particle_phase_space(), tbsize, dt);

	for (auto& i : work)
	{
		i();
	}
	work.clear();

	hd.planets_snapshot = HostPlanetSnapshot(hd.planets);

	t += dt * tbsize;
	step_and_upload_planets();
	hipStreamSynchronize(htd_stream);

	if (print_counter % print_every == 0)
	{
		double e;
		calculate_planet_metrics(hd.planets, &e, nullptr);

		double elapsed = time();
		double total = elapsed * (t_f - t_0) / (t - t_0);

		output << "t=" << t << " (" << elapsed / total * 100 << "% " << elapsed << "m elapsed, " << total << "m total " << total - elapsed << "m remain)" << std::endl;
		output << "Error = " << (e - e_0) / e_0 * 100 << ", " << dd.particle_phase_space().n_alive << " particles remaining" << std::endl;
	}
	print_counter++;

	if (timing_output)
	{
		double e_;
		f64_3 l_;
		calculate_planet_metrics(hd.planets, &e_, &l_);
	
		*timing_output << "ep " << e_ << std::endl;
		*timing_output << "lp " << l_.x << " " << l_.y << " " << l_.z << std::endl;
	}

	for (size_t i = hd.particles.n_alive; i < hd.particles.n_alive + hd.particles.n_encounter; i++)
	{
		// step_particle(..)
	}
	// memcy_htd(hd.n_alive, hd.n_encounter)
	// hd.n_alive = partition(...)
	// dd.n_alive = hd.n_alive

	hipStreamSynchronize(main_stream);
	resync();
}

void Executor::resync()
{
	auto& particles = dd.particle_phase_space();
	size_t prev_alive = particles.n_alive;

	particles.n_alive = thrust::stable_partition(thrust::cuda::par.on(par_stream), particles.begin(), particles.begin() + particles.n_alive, DeviceParticleUnflaggedPredicate())
		- particles.begin();
	hipStreamSynchronize(par_stream);

	size_t diff = prev_alive - particles.n_alive;

	ed = ExecutorData(diff);

	memcpy_dth(ed.r, particles.r, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.v, particles.v, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.id, particles.id, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.deathtime_index, particles.deathtime_index, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.deathflags, particles.deathflags, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);

	add_job([prev_alive, diff, this]()
		{
			std::unordered_map<size_t, size_t> indices;
			for (size_t i = 0; i < prev_alive; i++)
			{
				indices[hd.particles.id[i]] = i;
			}

			for (size_t i = 0; i < diff; i++)
			{
				size_t index = indices[ed.id[i]];
				hd.particles.r[index] = ed.r[i];
				hd.particles.v[index] = ed.v[i];
				hd.particles.deathflags[index] = ed.deathflags[i];

				std::ostream* output_stream = nullptr;

				if ((ed.deathflags[i] & 0x0001) && resolve_encounters)
				{
					output_stream = discard_output;
				}
				else
				{
					output_stream = discard_output;
					hd.particles.deathtime[index] = t + dt * ed.deathtime_index[i];
				}

				if (output_stream)
				{
					*output_stream << ed.r[i] << std::endl;
					*output_stream << ed.v[i] << std::endl;
					*output_stream << hd.particles.deathtime[index] << " " << ed.deathflags[i] << " " << ed.id[i] << std::endl;
					*output_stream << hd.planets.n - 1 << std::endl;
					for (size_t j = 1; j < hd.planets.n; j++)
					{
						*output_stream << hd.planets.m[j] << std::endl;
						*output_stream << hd.planets.r_log_slow[ed.deathtime_index[i] * (hd.planets.n - 1) + j - 1] << std::endl;
					}
				}
			}

			hd.particles.stable_partition_alive();
		});
}


void Executor::finish()
{
	hipStreamSynchronize(main_stream);
	resync();
	download_data();

	output << "Simulation finished. t = " << t << ". n_particle = " << hd.particles.n_alive << std::endl;
}
