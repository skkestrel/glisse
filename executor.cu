#include "hip/hip_runtime.h"
#include <iomanip>
#include <fstream>
#include <unordered_map>
#include <algorithm>

#include "types.h"
#include "executor.cuh"
#include "wh.cuh"
#include "convert.h"
#include "wh.h"

ExecutorData::ExecutorData() { }
ExecutorData::ExecutorData(size_t n)
{
	r = v = std::vector<f64_3>(n);
	id = deathtime_index = std::vector<uint32_t>(n);
	deathflags = std::vector<uint16_t>(n);
}

template<typename T>
hipError_t memcpy_dth(std::vector<T>& dest, const thrust::device_vector<T>& src, hipStream_t& stream, size_t destbegin = 0, size_t srcbegin = 0, size_t len = static_cast<uint32_t>(-1))
{
	if (len == static_cast<uint32_t>(-1))
	{
		len = src.size();
	}
	if (dest.size() < destbegin + len)
	{
		throw std::exception();
	}

	return hipMemcpyAsync(dest.data() + destbegin, src.data().get() + srcbegin, len * sizeof(T), hipMemcpyDeviceToHost, stream);
}

template<typename T>
hipError_t memcpy_htd(thrust::device_vector<T>& dest, const std::vector<T>& src, hipStream_t& stream, size_t destbegin = 0, size_t srcbegin = 0, size_t len = static_cast<uint32_t>(-1))
{
	if (len == static_cast<uint32_t>(-1))
	{
		len = src.size();
	}
	if (dest.size() < destbegin + len)
	{
		throw std::exception();
	}

	return hipMemcpyAsync(dest.data().get() + destbegin, src.data() + srcbegin, len * sizeof(T), hipMemcpyHostToDevice, stream);
}

struct DeviceParticleUnflaggedPredicate
{
	template<typename Tuple>
	__host__ __device__
	bool operator()(const Tuple& args)
	{
		uint8_t flag = thrust::get<3>(args);
		return flag == 0;
	}
};

Executor::Executor(HostData& hd, DeviceData& dd, std::ostream& out) : hd(hd), dd(dd), tbsize(128), ce_factor(1),
	output(out), resolve_encounters(false) { }

void Executor::init()
{
	to_helio(hd);
	initialize(hd.planets, hd.particles, wh_alloc);
	calculate_planet_metrics(hd.planets, wh_alloc, &e_0, nullptr);

	output << std::setprecision(7);
	output << "e_0 (planets) = " << e_0 << std::endl;
	output << "n_particle = " << hd.particles.n << std::endl;
	output << "n_particle_alive = " << hd.particles.n_alive << std::endl;
	output << "==================================" << std::endl;
	output << "Sending initial conditions to GPU." << std::endl;

	hipStreamCreate(&main_stream);
	hipStreamCreate(&htd_stream);
	hipStreamCreate(&par_stream);
	hipStreamCreate(&dth_stream);

	upload_data();
	output << "n_particle_alive = " << dd.particle_phase_space().n_alive << std::endl;

	resync();
	download_data();

	starttime = std::chrono::high_resolution_clock::now();

	output << "       Starting simulation.       " << std::endl << std::endl;

	if (encounter_output)
	{
		*encounter_output << std::setprecision(17);
	}

	step_and_upload_planets();
	if (!resolve_encounters)
	{
		ce_factor = 1;
	}
}

void Executor::step_and_upload_planets()
{
	if (resolve_encounters)
	{

		for (size_t i = 0; i < tbsize * ce_factor; i++)
		{
			step_planets(hd.planets, wh_alloc, t, i, dt / ce_factor);
			// take the planet positions at the end of every timestep

			if (i % ce_factor == ce_factor - 1)
			{
				size_t slow_index = i / ce_factor;

				auto fast_begin = hd.planets.r_log.begin() + i * (hd.planets.n - 1);
				std::copy(fast_begin, fast_begin + (hd.planets.n - 1), hd.planets.r_log_slow.begin() + slow_index * (hd.planets.n - 1));

				fast_begin = hd.planets.v_log.begin() + i * (hd.planets.n - 1);
				std::copy(fast_begin, fast_begin + (hd.planets.n - 1), hd.planets.v_log_slow.begin() + slow_index * (hd.planets.n - 1));

				hd.planets.h0_log_slow[i / ce_factor] = hd.planets.h0_log[i];
			}
		}
	}
	else
	{
		for (size_t i = 0; i < tbsize; i++)
		{
			step_planets(hd.planets, wh_alloc, t, i, dt);
		}

		std::copy(hd.planets.h0_log.begin(), hd.planets.h0_log.end(), hd.planets.h0_log_slow.begin());
		std::copy(hd.planets.r_log.begin(), hd.planets.r_log.end(), hd.planets.r_log_slow.begin());
		std::copy(hd.planets.v_log.begin(), hd.planets.v_log.end(), hd.planets.v_log_slow.begin());
	}

	// We only upload the planet log if any particles are going to use the planet log on the GPU
	// Cases where the planet log is not used by the particles:
	// - There are no particles alive on the GPU, AND there are no particles in close encounters on the CPU
	// since the particles that survive close encounters can make it to the GPU at the end of this timestep
	// and thus the next planet chunk will be required
	if (dd.particle_phase_space().n_alive == 0 && hd.particles.n_encounter == 0)
	{
		upload_planet_log();
	}
}

void Executor::upload_data()
{
	dd.particles0 = DeviceParticlePhaseSpace(hd.particles.n);
	dd.particles1 = DeviceParticlePhaseSpace(hd.particles.n);

	dd.planets0 = DevicePlanetPhaseSpace(hd.planets.n, tbsize);
	dd.planets1 = DevicePlanetPhaseSpace(hd.planets.n, tbsize);

	dd.planet_data_id = 0;
	dd.particle_data_id = 0;

	auto& particles = dd.particle_phase_space();

	particles.n_alive = hd.particles.n_alive;

	memcpy_htd(particles.r, hd.particles.r, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.v, hd.particles.v, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.a, hd.particles.a, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.deathflags, hd.particles.deathflags, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(particles.id, hd.particles.id, htd_stream);
	hipStreamSynchronize(htd_stream);

	memcpy_htd(dd.planet_phase_space().m, hd.planets.m, htd_stream);
	hipStreamSynchronize(htd_stream);

	dd.planet_data_id++;
	memcpy_htd(dd.planet_phase_space().m, hd.planets.m, htd_stream);
	hipStreamSynchronize(htd_stream);
}

void Executor::add_job(const std::function<void()>& job)
{
	work.push_back(std::move(job));
}

void Executor::download_data()
{
	auto& particles = dd.particle_phase_space();

	Vu32 prev_ids(hd.particles.id.begin(), hd.particles.id.end());

	memcpy_dth(hd.particles.r, particles.r, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.v, particles.v, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.a, particles.a, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.id, particles.id, dth_stream);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(hd.particles.deathflags, particles.deathflags, dth_stream);
	hipStreamSynchronize(dth_stream);

	// This should NEVER happen. I think this is a recoverable 
	// error, by swapping particle indices on the host, but that sounds annoying...
	if (prev_ids != hd.particles.id)
	{
		output << "WARNING! ID MISMATCH! WARNING!" << std::endl;
		throw std::exception();
	}

	hd.particles.n_alive = dd.particle_phase_space().n_alive;
}

void Executor::upload_planet_log()
{
	dd.planet_data_id++;
	auto& planets = dd.planet_phase_space();

	memcpy_htd(planets.h0_log, hd.planets.h0_log_slow, htd_stream);
	hipStreamSynchronize(htd_stream);
	memcpy_htd(planets.r_log, hd.planets.r_log_slow, htd_stream);
	hipStreamSynchronize(htd_stream);
}


double Executor::time() const
{
	auto now = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> millis = now - starttime;
	return millis.count() / 60000;
}

void Executor::loop()
{
	step_particles_cuda(main_stream, dd.planet_phase_space(), dd.particle_phase_space(), tbsize, dt);

	// The queued work should begin RIGHT after the CUDA call
	for (auto& i : work)
	{
		i();
	}
	work.clear();

	// The snapshot contains the planet states at the end of the previous timestep - 
	// consider removing this? We can use hd.planets.*_log_old[-1] to replicate this functionality
	hd.planets_snapshot = HostPlanetSnapshot(hd.planets);

	// The OLD logs are required by the close encounter handler
	std::swap(hd.planets.r_log, hd.planets.r_log_old);
	std::swap(hd.planets.v_log, hd.planets.v_log_old);
	std::swap(hd.planets.r_log_slow, hd.planets.r_log_slow_old);
	std::swap(hd.planets.v_log_slow, hd.planets.v_log_slow_old);

	std::swap(hd.planets.h0_log, hd.planets.h0_log_old);
	std::swap(hd.planets.h0_log_slow, hd.planets.h0_log_slow_old);

	t += dt * tbsize;
	step_and_upload_planets();
	hipStreamSynchronize(htd_stream);

	for (size_t i = hd.particles.n_alive; i < hd.particles.n_alive + hd.particles.n_encounter; i++)
	{
		// step_particle(..)
	}
	// hd.n_alive = hd.particles.stable_partition_alive(...)
	// memcy_htd(particles.n_alive, hd.n_encounter)
	// dd.n_alive = hd.n_alive

	hipStreamSynchronize(main_stream);
	resync();
}

void Executor::resync()
{
	// There's nothing to resync if all the particles on the device are dead!
	// Although dd.particles.n_alive can be out of sync with dd.particles.deathflags before
	// resync() is called, this is safe:
	// - The MVS kernel does not revive particles, so resync() will never INCREASE n_alive
	// - dd.particles.n_alive is adjusted by the close encounter handler just BEFORE this call
	if (dd.particle_phase_space().n_alive == 0) return;

	auto& particles = dd.particle_phase_space();
	size_t prev_alive = particles.n_alive;

	particles.n_alive = thrust::stable_partition(thrust::cuda::par.on(par_stream), particles.begin(), particles.begin() + particles.n_alive, DeviceParticleUnflaggedPredicate())
		- particles.begin();
	hipStreamSynchronize(par_stream);

	size_t diff = prev_alive - particles.n_alive;

	ed = ExecutorData(diff);

	memcpy_dth(ed.r, particles.r, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.v, particles.v, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.id, particles.id, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.deathtime_index, particles.deathtime_index, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);
	memcpy_dth(ed.deathflags, particles.deathflags, dth_stream, 0, particles.n_alive, diff);
	hipStreamSynchronize(dth_stream);

	std::unordered_map<size_t, size_t> indices;
	for (size_t i = 0; i < prev_alive; i++)
	{
		indices[hd.particles.id[i]] = i;
	}

	for (size_t i = 0; i < diff; i++)
	{
		size_t index = indices[ed.id[i]];
		hd.particles.r[index] = ed.r[i];
		hd.particles.v[index] = ed.v[i];
		hd.particles.deathflags[index] = ed.deathflags[i];

		if ((ed.deathflags[i] & 0x0001) && resolve_encounters)
		{
			// TODO do something here... maybe clear the death bit?
		}
		else
		{
			hd.particles.deathtime[index] = t - dt * (tbsize - ed.deathtime_index[i]);
		}
	}

	hd.particles.stable_partition_alive();
	hd.particles.n_encounter = hd.particles.n_alive - particles.n_alive;

	add_job([diff, this]()
		{
			for (size_t i = 0; i < diff; i++)
			{
				*encounter_output << ed.r[i] << std::endl;
				*encounter_output << ed.v[i] << std::endl;
				*encounter_output << ed.id[i] << " " << ed.deathflags[i] << " " << t - dt * (tbsize - ed.deathtime_index[i]) << std::endl;
				*encounter_output << hd.planets.n_alive << std::endl;

				*encounter_output << hd.planets.m[0] << std::endl;
				*encounter_output << f64_3(0) << std::endl;
				*encounter_output << f64_3(0) << std::endl;
				*encounter_output << hd.planets.id[0] << std::endl;
				for (size_t j = 1; j < hd.planets.n_alive; j++)
				{
					*encounter_output << hd.planets.m[j] << std::endl;
					*encounter_output << hd.planets.r_log_slow[ed.deathtime_index[i] * (hd.planets.n - 1) + j - 1] << std::endl;
					*encounter_output << hd.planets.v_log_slow[ed.deathtime_index[i] * (hd.planets.n - 1) + j - 1] << std::endl;
					*encounter_output << hd.planets.id[i] << std::endl;
				}
			}
		});
}


void Executor::finish()
{
	hipStreamSynchronize(main_stream);
	resync();
	download_data();

	output << "Simulation finished. t = " << t << ". n_particle = " << hd.particles.n_alive << std::endl;
}
