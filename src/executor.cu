#include "hip/hip_runtime.h"
#include <iomanip>
#include <fstream>
#include <unordered_map>
#include <algorithm>
#include <thread>

#include "util.cuh"
#include "util.h"
#include "types.h"
#include "executor.cuh"
#include "wh.cuh"
#include "convert.h"

namespace sr
{
namespace exec
{
	using namespace sr::wh;
	using namespace sr::util;
	using namespace sr::convert;
	using namespace sr::data;

	ExecutorData::ExecutorData() { }
	ExecutorData::ExecutorData(size_t n)
	{
		r = v = std::vector<f64_3>(n);
		deathflags = std::vector<uint16_t>(n);
		id = std::vector<uint32_t>(n);
		deathtime_index = std::vector<uint32_t>(n);
	}

	struct DeviceParticleUnflaggedPredicate
	{
		template<typename Tuple>
		__host__ __device__
		bool operator()(const Tuple& args)
		{
			uint16_t flag = thrust::get<2>(thrust::get<0>(args));
			return flag == 0;
		}
	};

	Executor::Executor(HostData& _hd, DeviceData& _dd, const Configuration& _config, std::ostream& out)
		: hd(_hd), dd(_dd), output(out), config(_config) { }

	void Executor::init()
	{
		if (!config.use_gpu)
		{
			output << "Executable was compiled with CUDA but USE_GPU was disabled!" << std::endl;
			throw std::exception();
		}

		to_helio(hd);

		integrator = sr::wh::WHCudaIntegrator(hd.planets, hd.particles, config);
		calculate_planet_metrics(hd.planets, &e_0, nullptr);

		output << std::setprecision(7);
		output << "e_0 (planets) = " << e_0 << std::endl;
		output << "n_particle = " << hd.particles.n() << std::endl;
		output << "n_particle_alive = " << hd.particles.n_alive() << std::endl;
		output << "==================================" << std::endl;
		output << "Sending initial conditions to GPU." << std::endl;

		hipStreamCreate(&main_stream);
		hipStreamCreate(&htd_stream);
		hipStreamCreate(&par_stream);
		hipStreamCreate(&dth_stream);

		hipEventCreate(&start_event);
		hipEventCreate(&cpu_finish_event);
		hipEventCreate(&gpu_finish_event);

		dd.particles = DeviceParticlePhaseSpace(hd.particles.n());

		dd.planets0 = DevicePlanetPhaseSpace(hd.planets.n(), config.tbsize);
		dd.planets1 = DevicePlanetPhaseSpace(hd.planets.n(), config.tbsize);
		dd.planet_data_id = 0;

		memcpy_htd(dd.planet_phase_space().m, hd.planets.m(), htd_stream);
		hipStreamSynchronize(htd_stream);
		dd.planet_data_id++;
		memcpy_htd(dd.planet_phase_space().m, hd.planets.m(), htd_stream);
		hipStreamSynchronize(htd_stream);

		if (hd.particles.n() > 0)
		{
			upload_data(0, hd.particles.n());
		}

		download_data();

		starttime = std::chrono::high_resolution_clock::now();

		output << "       Starting simulation.       " << std::endl << std::endl;

		if (encounter_output)
		{
			*encounter_output << std::setprecision(17);
		}

		step_and_upload_planets();
	}

	void Executor::swap_logs()
	{
		hd.planets.swap_logs();
		integrator.swap_logs();
	}

	void Executor::step_and_upload_planets()
	{
		integrator.integrate_planets_timeblock(hd.planets, t);

		swap_logs();

		// We only upload the planet log if any particles are going to use the planet log on the GPU
		// Cases where the planet log is not used by the particles:
		// - There are no particles alive on the GPU, AND there are no particles in close encounters on the CPU
		// since the particles that survive close encounters can make it to the GPU at the end of this timestep
		// and thus the next planet chunk will be required

		if (dd.particle_phase_space().n_alive > 0 || hd.particles.n_encounter() > 0)
		{
			upload_planet_log();
		}
	}

	void Executor::upload_data(size_t begin, size_t length)
	{
		auto& particles = dd.particle_phase_space();
		particles.n_alive = hd.particles.n_alive();
		integrator.upload_data_cuda(htd_stream, begin, length);

		memcpy_htd(particles.r, hd.particles.r(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
		memcpy_htd(particles.v, hd.particles.v(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
		memcpy_htd(particles.deathflags, hd.particles.deathflags(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
		memcpy_htd(particles.id, hd.particles.id(), htd_stream, begin, begin, length);
		hipStreamSynchronize(htd_stream);
	}

	void Executor::add_job(const std::function<void()>& job)
	{
		work.push_back(std::move(job));
	}

	void Executor::download_data(bool ignore_errors)
	{
		auto& particles = dd.particle_phase_space();

		Vu32 prev_ids(hd.particles.id().begin(), hd.particles.id().end());

		memcpy_dth(hd.particles.r(), particles.r, dth_stream, 0, 0, particles.n_alive);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(hd.particles.v(), particles.v, dth_stream, 0, 0, particles.n_alive);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(hd.particles.id(), particles.id, dth_stream, 0, 0, particles.n_alive);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(hd.particles.deathflags(), particles.deathflags, dth_stream, 0, 0, particles.n_alive);
		hipStreamSynchronize(dth_stream);

		// This should NEVER happen. I think this is a recoverable 
		// error, by swapping particle indices on the host, but that sounds annoying...
		if (prev_ids != hd.particles.id())
		{
			output << "WARNING! ID MISMATCH! WARNING!" << std::endl;

			if (!ignore_errors)
			{
				throw std::exception();
			}
		}

		hd.particles.n_alive() = dd.particle_phase_space().n_alive;
	}

	void Executor::upload_planet_log()
	{
		dd.planet_data_id++;
		auto& planets = dd.planet_phase_space();

		memcpy_htd(planets.r_log, hd.planets.r_log().slow, htd_stream);
		hipStreamSynchronize(htd_stream);

		integrator.upload_planet_log_cuda(htd_stream, dd.planet_data_id);
	}


	double Executor::time() const
	{
		auto now = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> millis = now - starttime;
		return millis.count() / 60000;
	}

	void Executor::loop(double* cputimeout, double* gputimeout)
	{
		std::thread cpu_thread;
		
		if (dd.particle_phase_space().n_alive > 0)
		{
			hipEventRecord(start_event, main_stream);
			integrator.integrate_particles_timeblock_cuda(main_stream, dd.planet_data_id, dd.planet_phase_space(), dd.particle_phase_space());
			hipEventRecord(gpu_finish_event, main_stream);
		}

		// The queued work should begin RIGHT after the CUDA call
		for (auto& i : work) i();
		work.clear();


		size_t encounter_start = hd.particles.n_alive() - hd.particles.n_encounter();
		for (size_t i = encounter_start; i < hd.particles.n_alive(); i++)
		{
			integrator.integrate_encounter_particle_catchup(hd.planets, hd.particles, i,
					ed.deathtime_index[i - encounter_start],
					t - config.dt * static_cast<double>(config.tbsize - ed.deathtime_index[i - encounter_start])
				);
		}

		auto gather_indices = hd.particles.stable_partition_alive(encounter_start, hd.particles.n_encounter());
		integrator.gather_particles(*gather_indices, encounter_start, hd.particles.n_encounter());
		upload_data(encounter_start, hd.particles.n_encounter());

		// Fill deathtime index with 0 so that the continuation will work
		thrust::fill(thrust::cuda::par.on(htd_stream), dd.particles.deathtime_index.begin() + encounter_start,
				dd.particles.deathtime_index.begin() + encounter_start + hd.particles.n_encounter(), 0);

		// The snapshot contains the planet states at the end of the previous timestep - 
		// consider removing this? We can use hd.planets.*_log_old()[-1] to replicate this functionality

		// Copy assignment ctor
		hd.planets_snapshot = hd.planets.base;

		t += config.dt * static_cast<double>(config.tbsize);
		step_and_upload_planets();

		if (dd.particle_phase_space().n_alive > 0)
		{
			hipStreamSynchronize(htd_stream);
			hipEventRecord(cpu_finish_event, par_stream);
			hipEventSynchronize(gpu_finish_event);

			float cputime, gputime;
			hipEventElapsedTime(&cputime, start_event, cpu_finish_event);
			hipEventElapsedTime(&gputime, start_event, gpu_finish_event);
			if (cputimeout) *cputimeout = cputime;
			if (gputimeout) *gputimeout = gputime;

			// There's nothing to resync if all the particles on the device are dead!
			// Although dd.particles.n_alive can be out of sync with dd.particles.deathflags before
			// resync() is called, this is safe:
			// - The MVS kernel does not revive particles, so resync() will never INCREASE n_alive
			// - dd.particles.n_alive is adjusted by the close encounter handler just BEFORE this call
			resync();
		}
	}

	void Executor::resync()
	{
		auto& particles = dd.particle_phase_space();
		size_t prev_alive = particles.n_alive;

		auto partition_it = thrust::make_zip_iterator(thrust::make_tuple(particles.begin(), integrator.device_begin()));
		particles.n_alive = thrust::stable_partition(thrust::cuda::par.on(main_stream),
				partition_it, partition_it + particles.n_alive, DeviceParticleUnflaggedPredicate()) - partition_it;
		hipStreamSynchronize(main_stream);

		size_t diff = prev_alive - particles.n_alive;

		ed = ExecutorData(diff);

		memcpy_dth(ed.r, particles.r, dth_stream, 0, particles.n_alive, diff);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(ed.v, particles.v, dth_stream, 0, particles.n_alive, diff);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(ed.id, particles.id, dth_stream, 0, particles.n_alive, diff);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(ed.deathtime_index, particles.deathtime_index, dth_stream, 0, particles.n_alive, diff);
		hipStreamSynchronize(dth_stream);
		memcpy_dth(ed.deathflags, particles.deathflags, dth_stream, 0, particles.n_alive, diff);
		hipStreamSynchronize(dth_stream);

		for (size_t i = 0; i < diff; i++)
		{
			if ((ed.deathflags[i] & 0x00FF) == 0x0001)
			{
				if (config.resolve_encounters)
				{
				}
				else
				{
					// If encounters are not being dealt with, kill the particle!
					ed.deathflags[i] |= 0x0080;
				}
			}
		}

		std::unique_ptr<std::vector<size_t>> ed_indices;
		stable_partition_alive_indices(ed.deathflags, 0, diff, &ed_indices);
		gather(ed.r, *ed_indices, 0, diff);
		gather(ed.v, *ed_indices, 0, diff);
		gather(ed.id, *ed_indices, 0, diff);
		gather(ed.deathflags, *ed_indices, 0, diff);
		gather(ed.deathtime_index, *ed_indices, 0, diff);

		std::unordered_map<size_t, size_t> indices;
		for (size_t i = 0; i < prev_alive; i++)
		{
			indices[hd.particles.id()[i]] = i;
		}

		for (size_t i = 0; i < diff; i++)
		{
			size_t index = indices[ed.id[i]];
			hd.particles.r()[index] = ed.r[i];
			hd.particles.v()[index] = ed.v[i];
			hd.particles.deathflags()[index] = ed.deathflags[i];

			if (ed.deathflags[i])
			{
				hd.particles.deathtime()[index] = static_cast<float>(t - config.dt * static_cast<double>(config.tbsize - ed.deathtime_index[i]));
			}
		}

		auto gather_indices = hd.particles.stable_partition_alive(0, prev_alive);
		integrator.gather_particles(*gather_indices, 0, prev_alive);

		hd.particles.n_encounter() = hd.particles.n_alive() - particles.n_alive;

		size_t encounter_start = particles.n_alive;

		add_job([encounter_start, diff, this]()
			{
				if (encounter_output)
				{
					for (size_t i = hd.particles.n_encounter(); i < diff; i++)
					{
						*encounter_output << hd.particles.r()[encounter_start + i] << std::endl;
						*encounter_output << hd.particles.v()[encounter_start + i] << std::endl;
						*encounter_output << hd.particles.id()[encounter_start + i] << " "
							<< hd.particles.deathflags()[encounter_start + i] << " "
							<< t - config.dt * static_cast<double>(config.tbsize - ed.deathtime_index[i]) << " death"
							<< std::endl;
						*encounter_output << hd.planets.n_alive() << std::endl;

						*encounter_output << hd.planets.m()[0] << std::endl;
						*encounter_output << f64_3(0) << std::endl;
						*encounter_output << f64_3(0) << std::endl;
						*encounter_output << hd.planets.id()[0] << std::endl;
						for (size_t j = 1; j < hd.planets.n_alive(); j++)
						{
							*encounter_output << hd.planets.m()[j] << std::endl;
							*encounter_output << hd.planets.r_log().slow[ed.deathtime_index[i] * (hd.planets.n() - 1) + j - 1] << std::endl;
							*encounter_output << hd.planets.v_log().slow[ed.deathtime_index[i] * (hd.planets.n() - 1) + j - 1] << std::endl;
							*encounter_output << hd.planets.id()[j] << std::endl;
						}
					}

					*encounter_output << std::flush;
				}
			});
	}


	void Executor::finish()
	{
		hipStreamSynchronize(main_stream);

		for (auto& i : work) i();
		work.clear();

		resync();

		for (auto& i : work) i();
		work.clear();

		output << "Simulation finished. t = " << t << ". n_particle = " << hd.particles.n_alive() << std::endl;
	}
}
}
