#include "hip/hip_runtime.h"
#include "types.cuh"
#include "wh.cuh"
#include "convert.h"
#include "util.cuh"

namespace sr
{
namespace wh
{
	using namespace sr::data;

	const size_t MAXKEP = 5;
	const float64_t TOLKEP = 1E-14;

	struct MVSKernel
	{
		const float64_t* planet_m;
		const float64_t mu;
		const f64_3* planet_h0_log;
		const f64_3* planet_r_log;
		const size_t planet_n;
		const size_t tbsize;
		const float64_t dt;

		const float64_t r2;
		const float64_t* planet_rh;

		MVSKernel(float64_t mu_, const f64_3* planet_r_log_, const float64_t* planet_m_, const size_t planet_n_, const f64_3* h0_log, const float64_t* _planet_rh, double _r2, size_t _tbsize, float64_t _dt) :
			planet_m(planet_m_),
			mu(mu_),
			planet_h0_log(h0_log),
			planet_r_log(planet_r_log_),
			planet_n(planet_n_),
			tbsize(_tbsize),
			dt(_dt),
			r2(_r2),
			planet_rh(_planet_rh)
		{ }

		__host__ __device__
		void kepeq(double dM, double ecosEo, double esinEo, double* dE, double* sindE, double* cosdE, uint16_t& flags) const
		{
			double f, fp, delta;

			*sindE = sin(*dE);
			*cosdE = cos(*dE);

			for (size_t i = 0; i < MAXKEP; i++)
			{
				f = *dE - ecosEo * (*sindE) + esinEo * (1. - *cosdE) - dM;
				fp = 1. - ecosEo * (*cosdE) + esinEo * (*sindE);
				delta = -f / fp;

				*dE += delta;
				*sindE = sin(*dE);
				*cosdE = cos(*dE);
			}

			flags = static_cast<uint16_t>(flags | ((fabs(delta) > TOLKEP) << 3));
		}

		__host__ __device__
		void drift(f64_3& r, f64_3& v, uint16_t& flags) const
		{
			float64_t dist = sqrt(r.lensq());
			float64_t vdotr = v.x * r.x + v.y * r.y + v.z * r.z;

			float64_t energy = v.lensq() * 0.5 - mu / dist;

			flags = static_cast<uint16_t>(flags | ((energy >= 0) << 2));

			float64_t a = -0.5 * mu / energy;
			float64_t n_ = sqrt(mu / (a * a * a));
			float64_t ecosEo = 1.0 - dist / a;
			float64_t esinEo = vdotr / (n_ * a * a);
			// float64_t e = sqrt(ecosEo * ecosEo + esinEo * esinEo);

			// subtract off an integer multiple of complete orbits
			float64_t dM = this->dt * n_ - M_2PI * (int) (dt * n_ / M_2PI);

			// remaining time to advance
			float64_t _dt = dM / n_;

			// call kepler equation solver with initial guess in dE already
			float64_t dE = dM - esinEo + esinEo * cos(dM) + ecosEo * sin(dM);
			float64_t sindE, cosdE;
			kepeq(dM, ecosEo, esinEo, &dE, &sindE, &cosdE, flags);

			float64_t fp = 1.0 - ecosEo * cosdE + esinEo * sindE;
			float64_t f = 1.0 + a * (cosdE - 1.0) / dist;
			float64_t g = _dt + (sindE - dE) / n_;
			float64_t fdot = -n_ * sindE * a / (dist * fp);
			float64_t gdot = 1.0 + (cosdE - 1.0) / fp;

			f64_3 r0 = r;
			r = r0 * f + v * g;
			v = r0 * fdot + v * gdot;
		}

		template<typename Tuple>
		__host__ __device__
		void operator()(Tuple args) const
		{
			f64_3 r = thrust::get<0>(thrust::get<0>(args));
			f64_3 v = thrust::get<1>(thrust::get<0>(args));
			uint16_t flags = thrust::get<2>(thrust::get<0>(args));

			uint32_t deathtime_index = 0;
			f64_3 a = thrust::get<1>(args);

			size_t _tbsize = this->tbsize;
			const f64_3* h0_log = this->planet_h0_log;
			const f64_3* r_log = this->planet_r_log;
			const float64_t* m = this->planet_m;
			const float64_t* rh = this->planet_rh;
			float64_t _r2 = this->r2;
			float64_t _dt = this->dt;

			for (uint32_t step = 0; step < static_cast<uint32_t>(_tbsize); step++)
			{
				if (flags == 0)
				{
					if (step < 100)
					{
					printf("%.26f %.26f %.26f\n", r.x, r.y, r.z);
					printf("%.26f %.26f %.26f\n", v.x, v.y, v.z);
					printf("%.26f %.26f %.26f\n\n", a.x, a.y, a.z);
					}

					// kick
					v = v + a * (_dt / 2);

					drift(r, v, flags);

					a = h0_log[step];

					// planet 0 is not counted
					for (uint32_t i = 1; i < static_cast<uint32_t>(planet_n); i++)
					{
						f64_3 dr = r - *(r_log + step * (planet_n - 1) + i - 1);
						if (step < 100)
						printf("%.26f %.26f %.26f\n", dr.x, dr.y, dr.z);

						float64_t rad = dr.x * dr.x;
						rad += dr.y * dr.y;
						rad += dr.z * dr.z;

						if (step < 100)
						printf("%.26f\n", rad);

						if (rad < rh[i] * rh[i] * _r2 * _r2 && flags == 0)
						{
							flags = flags & 0x00FF;
							flags = static_cast<uint16_t>(flags | (i << 8) | 0x0001);
						}

						float64_t inv3 = 1. / (rad * sqrt(rad));
						float64_t fac = m[i] * inv3;

						a -= dr * fac;

						if (step < 100)
						printf("%.26f %.26f %.26f\n\n", a.x, a.y, a.z);
					}

					float64_t rad = r.lensq();
					if (rad < rh[0] * rh[0] * _r2 * _r2)
					{
						flags = flags & 0x00FF;
						flags = flags | 0x0001;
					}
					if (rad > 200 * 200)
					{
						flags = flags | 0x0002;
					}


					v = v + a * (_dt / 2);

					deathtime_index = step + 1;
				}
			}

			thrust::get<0>(thrust::get<0>(args)) = r;
			thrust::get<1>(thrust::get<0>(args)) = v;
			thrust::get<2>(thrust::get<0>(args)) = flags;
			thrust::get<3>(thrust::get<0>(args)) = deathtime_index;

			thrust::get<1>(args) = a;
		}
	};

	WHCudaIntegrator::WHCudaIntegrator() { }

	WHCudaIntegrator::WHCudaIntegrator(HostPlanetPhaseSpace& pl, HostParticlePhaseSpace& pa, const Configuration& config)
		: base(pl, pa, config)
	{
		device_h0_log_0 = Dvf64_3(config.tbsize);
		device_h0_log_1 = Dvf64_3(config.tbsize);
		device_particle_a = Dvf64_3(pa.n());

		device_planet_rh = Dvf64(pl.n());

		memcpy_htd(device_planet_rh, base.planet_rh, 0);
		hipStreamSynchronize(0);
	}

	Dvf64_3& WHCudaIntegrator::device_h0_log(size_t planet_data_id)
	{
		return planet_data_id % 2 ? device_h0_log_1 : device_h0_log_0;
	}

	void WHCudaIntegrator::upload_planet_log_cuda(hipStream_t stream, size_t planet_data_id)
	{
		memcpy_htd(device_h0_log(planet_data_id), base.planet_h0_log.slow, stream);
		hipStreamSynchronize(stream);
	}

	void WHCudaIntegrator::gather_particles(const std::vector<size_t>& indices, size_t begin, size_t length)
	{
		base.gather_particles(indices, begin, length);
	}

	void WHCudaIntegrator::integrate_planets_timeblock(HostPlanetPhaseSpace& pl, float64_t t)
	{
		base.integrate_planets_timeblock(pl, t);
	}

	void WHCudaIntegrator::integrate_particles_timeblock(const HostPlanetPhaseSpace& pl, HostParticlePhaseSpace& pa, size_t begin, size_t length, float64_t t)
	{
		base.integrate_particles_timeblock(pl, pa, begin, length, t);
	}

	void WHCudaIntegrator::integrate_encounter_particle_catchup(const HostPlanetPhaseSpace& pl, HostParticlePhaseSpace& pa, size_t particle_index, size_t particle_deathtime_index, double t)
	{
		base.integrate_encounter_particle_catchup(pl, pa, particle_index, particle_deathtime_index, t);
	}

	void WHCudaIntegrator::swap_logs()
	{
		base.swap_logs();
	}

	void WHCudaIntegrator::upload_data_cuda(hipStream_t stream, size_t begin, size_t length)
	{
		memcpy_htd(device_particle_a, base.particle_a, stream, begin, begin, length);
		hipStreamSynchronize(stream);
	}

	void WHCudaIntegrator::integrate_particles_timeblock_cuda(hipStream_t stream, const HostPlanetPhaseSpace& pl_h, size_t planet_data_id, const DevicePlanetPhaseSpace& pl, DeviceParticlePhaseSpace& pa)
	{
		auto it = thrust::make_zip_iterator(thrust::make_tuple(pa.begin(), device_begin()));

		thrust::for_each(thrust::cuda::par.on(stream), it, it + pa.n_alive,
			MVSKernel(
				pl_h.m()[0],
				pl.r_log.data().get(),
				pl.m.data().get(),
				pl.n_alive,
				device_h0_log(planet_data_id).data().get(),
				device_planet_rh.data().get(),
				base.encounter_r2,
				base.tbsize, base.dt));
	}
}
}
