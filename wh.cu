#include "hip/hip_runtime.h"
#include "types.cuh"
#include "wh.cuh"
#include "convert.h"
#include "util.cuh"

const size_t MAXKEP = 5;
const float64_t TOLKEP = 1E-14;

struct MVSKernel
{
	const float64_t* planet_m;
	const float64_t mu;
	const f64_3* planet_h0_log;
	const f64_3* planet_r_log;
	const size_t planet_n;
	const size_t tbsize;
	const float64_t dt;

	const float64_t r2;
	const float64_t* planet_rh;

	MVSKernel(const DevicePlanetPhaseSpace& planets, const Dvf64_3& h0_log, const Dvf64& planet_rh, double r2, size_t tbsize, float64_t dt)
		: planet_m(planets.m.data().get()), mu(planets.m[0]), planet_h0_log(h0_log.data().get()),
		planet_r_log(planets.r_log.data().get()), planet_n(planets.n_alive), tbsize(tbsize), dt(dt),
		r2(r2), planet_rh(planet_rh.data().get())
	{ }

	__host__ __device__
	void kepeq(double dM, double ecosEo, double esinEo, double* dE, double* sindE, double* cosdE, uint16_t& flags) const
	{
		double f, fp, delta;

		*sindE = sin(*dE);
		*cosdE = cos(*dE);

		for (size_t i = 0; i < MAXKEP; i++)
		{
			f = *dE - ecosEo * (*sindE) + esinEo * (1. - *cosdE) - dM;
			fp = 1. - ecosEo * (*cosdE) + esinEo * (*sindE);
			delta = -f / fp;

			*dE += delta;
			*sindE = sin(*dE);
			*cosdE = cos(*dE);
		}

		flags = static_cast<uint16_t>(flags | ((fabs(delta) > TOLKEP) << 3));
	}

	__host__ __device__
	void drift(f64_3& r, f64_3& v, uint16_t& flags) const
	{
		float64_t dist = sqrt(r.lensq());
		float64_t vdotr = v.x * r.x + v.y * r.y + v.z * r.z;

		float64_t energy = v.lensq() * 0.5 - mu / dist;

		flags = static_cast<uint16_t>(flags | ((energy >= 0) << 2));

		float64_t a = -0.5 * mu / energy;
		float64_t n_ = sqrt(mu / (a * a * a));
		float64_t ecosEo = 1.0 - dist / a;
		float64_t esinEo = vdotr / (n_ * a * a);
		float64_t e = sqrt(ecosEo * ecosEo + esinEo * esinEo);

		// subtract off an integer multiple of complete orbits
		float64_t dM = this->dt * n_ - M_2PI * (int) (dt * n_ / M_2PI);

		// remaining time to advance
		float64_t dt = dM / n_;

		// call kepler equation solver with initial guess in dE already
		float64_t dE = dM - esinEo + esinEo * cos(dM) + ecosEo * sin(dM);
		float64_t sindE, cosdE;
		kepeq(dM, ecosEo, esinEo, &dE, &sindE, &cosdE, flags);

		float64_t fp = 1.0 - ecosEo * cosdE + esinEo * sindE;
		float64_t f = 1.0 + a * (cosdE - 1.0) / dist;
		float64_t g = dt + (sindE - dE) / n_;
		float64_t fdot = -n_ * sindE * a / (dist * fp);
		float64_t gdot = 1.0 + (cosdE - 1.0) / fp;

		f64_3 r0 = r;
		r = r0 * f + v * g;
		v = r0 * fdot + v * gdot;
	}

	template<typename Tuple>
	__host__ __device__
	void operator()(Tuple args) const
	{
		f64_3 r = thrust::get<0>(thrust::get<0>(args));
		f64_3 v = thrust::get<1>(thrust::get<0>(args));
		uint16_t flags = thrust::get<2>(thrust::get<0>(args));

		uint32_t deathtime_index = 0;
		f64_3 a = thrust::get<1>(args);

		size_t tbsize = this->tbsize;
		const f64_3* h0_log = this->planet_h0_log;
		const f64_3* r_log = this->planet_r_log;
		const float64_t* m = this->planet_m;
		const float64_t* rh = this->planet_rh;
		float64_t r2 = this->r2;
		float64_t dt = this->dt;

		for (uint32_t step = 0; step < static_cast<uint32_t>(tbsize); step++)
		{
			if (flags == 0)
			{
				// kick
				v = v + a * (dt / 2);

				drift(r, v, flags);

				a = h0_log[step];

				// planet 0 is not counted
				for (uint32_t i = 1; i < static_cast<uint32_t>(planet_n); i++)
				{
					f64_3 dr = r - r_log[step * (planet_n - 1) + i - 1];

					float64_t rad = dr.lensq();

					if (rad < rh[i] * rh[i] * r2 * r2 && flags == 0)
					{
						flags = static_cast<uint16_t>(flags | (i << 8) | 0x0001);
					}

					float64_t inv3 = 1. / (rad * sqrt(rad));
					float64_t fac = m[i] * inv3;

					a -= dr * fac;
				}

				float64_t rad = r.lensq();
				if (rad < rh[0] * rh[0] * r2 * r2)
				{
					flags = flags | 0x0001;
				}
				if (rad > 200 * 200)
				{
					flags = flags | 0x0002;
				}


				v = v + a * (dt / 2);

				deathtime_index = step + 1;
			}
		}

		thrust::get<0>(thrust::get<0>(args)) = r;
		thrust::get<1>(thrust::get<0>(args)) = v;
		thrust::get<2>(thrust::get<0>(args)) = flags;
		thrust::get<3>(thrust::get<0>(args)) = deathtime_index;

		thrust::get<1>(args) = a;
	}
};

WHCudaIntegrator::WHCudaIntegrator() { }

WHCudaIntegrator::WHCudaIntegrator(HostPlanetPhaseSpace& pl, HostParticlePhaseSpace& pa, const Configuration& config)
	: base(pl, pa, config)
{
	device_h0_log_0 = Dvf64_3(config.tbsize);
	device_h0_log_1 = Dvf64_3(config.tbsize);
	device_particle_a = Dvf64_3(pa.n);

	device_planet_rh = Dvf64(pl.n);

	memcpy_htd(device_planet_rh, base.planet_rh, 0);
	hipStreamSynchronize(0);
}

Dvf64_3& WHCudaIntegrator::device_h0_log(size_t planet_data_id)
{
	return planet_data_id % 2 ? device_h0_log_1 : device_h0_log_0;
}

void WHCudaIntegrator::upload_planet_log_cuda(hipStream_t stream, size_t planet_data_id)
{
	memcpy_htd(device_h0_log(planet_data_id), base.planet_h0_log.get<true, false>(), stream);
	hipStreamSynchronize(stream);
}

void WHCudaIntegrator::gather_particles(const std::vector<size_t>& indices, size_t begin, size_t length)
{
	base.gather_particles(indices, begin, length);
}

void WHCudaIntegrator::integrate_planets_timeblock(HostPlanetPhaseSpace& pl, float64_t t)
{
	base.integrate_planets_timeblock(pl, t);
}

void WHCudaIntegrator::integrate_particles_timeblock(const HostPlanetPhaseSpace& pl, HostParticlePhaseSpace& pa, size_t begin, size_t length, float64_t t)
{
	base.integrate_particles_timeblock(pl, pa, begin, length, t);
}

void WHCudaIntegrator::integrate_encounter_particle_catchup(const HostPlanetPhaseSpace& pl, HostParticlePhaseSpace& pa, size_t particle_index, size_t particle_deathtime_index, size_t planet_index)
{
	base.integrate_encounter_particle_catchup(pl, pa, particle_index, particle_deathtime_index, planet_index);
}

void WHCudaIntegrator::upload_data_cuda(hipStream_t stream, size_t begin, size_t length)
{
	memcpy_htd(device_particle_a, base.particle_a, stream, begin, begin, length);
	hipStreamSynchronize(stream);
}

void WHCudaIntegrator::integrate_particles_timeblock_cuda(hipStream_t stream, size_t planet_data_id, const DevicePlanetPhaseSpace& pl, DeviceParticlePhaseSpace& pa)
{
	if (pa.n_alive > 0)
	{
		auto it = thrust::make_zip_iterator(thrust::make_tuple(pa.begin(), device_begin()));
		thrust::for_each(thrust::cuda::par.on(stream), it, it + pa.n_alive, MVSKernel(pl, device_h0_log(planet_data_id), device_planet_rh, base.encounter_r2, base.tbsize, base.dt));
	}
}
